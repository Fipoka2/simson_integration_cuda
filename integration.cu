#include "hip/hip_runtime.h"
#include "integration.h"
#include <math_functions.h>
#include <math.h>

namespace {
    const unsigned int MAX_THREADS = 1024;
    const unsigned int MAX_BLOCKS = 2147483647;
};

__constant__  float LEFT;
__constant__  float RIGHT;
__constant__  int SEGMENTS;
__constant__  float STEP;

float formula(float x) {
    return log10f(x);
};


__global__ void reduce(float *g_odata) {

    __shared__ float sdata[1024];

    // each thread loads one element from global to shared mem
    // note use of 1D thread indices (only) in this kernel
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i >= SEGMENTS) {
        sdata[threadIdx.x] = 0;
    }
    if (i < SEGMENTS ) {

        float val = log10f(LEFT + STEP * (i+1));
        sdata[threadIdx.x] =  (i+1)%2 == 0 ? 2 * val : 4 * val;

        __syncthreads();
        // do reduction in shared mem
        for (int s=1; s < blockDim.x; s *=2)
        {
            int index = 2 * s * threadIdx.x;

            if (index < blockDim.x)
            {
                sdata[index] += sdata[index + s];
            }
            __syncthreads();
        }

        // write result for this block to global mem
        if (threadIdx.x == 0)
            atomicAdd(g_odata,sdata[0]);
    }
}

Result integrateOnGpu(float left, float right, int segments, float step) {

    hipEvent_t start, stop;
    hipEventCreate ( &start );
    hipEventCreate ( &stop );
    hipEventRecord ( start, 0 );

    float* dev_result;
//    int seg1 = segments-1;

    hipMalloc((void **) &dev_result, sizeof(float));
//    hipMemcpyToSymbol(HIP_SYMBOL(LEFT), &left, sizeof(float));
//    hipMemcpyToSymbol(HIP_SYMBOL(RIGHT), &right, sizeof(float));
//    hipMemcpyToSymbol(HIP_SYMBOL(SEGMENTS), &seg1, sizeof(int));
//    hipMemcpyToSymbol(HIP_SYMBOL(STEP), &step, sizeof(float));
    hipSetDevice(0);
    const unsigned int numBlocks = (segments-1) / MAX_THREADS + 1;
    reduce <<< numBlocks, MAX_THREADS >>> (dev_result);

    float* result = new float(0);

    hipMemcpy(result, dev_result, sizeof(float) , hipMemcpyDeviceToHost);
    hipFree(dev_result);
    hipEventRecord ( stop, 0 );
    hipEventSynchronize ( stop );
    hipDeviceSynchronize();

    Result r;
    hipEventElapsedTime ( &(r.time), start, stop );
    r.value = (step / 3) * (formula(left) + formula(right) + (*result));

    free(result);
    return r;
}

void setGPUConstants (float left, float right, int segments, float step) {
    int seg1 = segments-1;

    hipMemcpyToSymbol(HIP_SYMBOL(LEFT), &left, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(RIGHT), &right, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(SEGMENTS), &seg1, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(STEP), &step, sizeof(float));
}


